
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a,int N)
{
    int i;
    for(i=0;i<N;++i)
    {
        a[i]=i;
    }
}

__global__ void doubleElements(int *a,int N)
{
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N)
    {
        a[i]*=2;
    }
}

bool checkElements(int*a,int N)
{
    int i;
    for(i=0;i<N;++i)
    {
        if(a[i]!=i*2) return false;
    }
    return true;

}

int main()
{
    int N=1000;
    int *a;
    size_t  size=N*sizeof(int);
    hipMallocManaged(&a,size);
    init(a,N);
//    size_t thread_per_block=10;
//    size_t number_of_block=10;
    size_t thread_per_block=256;
    size_t number_of_block=(N+thread_per_block-1)/thread_per_block;
    doubleElements<<<thread_per_block,number_of_block>>>(a,N);
    hipDeviceSynchronize();
    bool aredouble=checkElements(a,N);
    printf("are all elements true? %s\n",aredouble?"true":"false");
    hipFree(a);

}