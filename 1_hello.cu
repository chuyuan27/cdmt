
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
    printf("hello from CPU.\n");
}

__global__ void helloGPU()
{
    printf("hello also from CPU.\n");
}

int main()
{
    helloCPU();
    helloGPU<<<1,2>>>();
    hipDeviceSynchronize();


}
