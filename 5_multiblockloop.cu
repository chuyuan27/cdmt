
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiblockloop()
{
    int c;
    c = threadIdx.x + blockIdx.x * blockDim.x;
    printf("%d\n",c);

}

int main()
{
    multiblockloop<<<10,10>>>();
    hipDeviceSynchronize();

}