
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void singleblockloop()
{
    int N = 10;
    for(int i = 0 ; i < N ; ++i)
    {
        if(threadIdx.x==i)
        {
            printf("%d",i);
        }

    }
}

int main()
{
    singleblockloop<<<1,10>>>();
    hipDeviceSynchronize();

}