
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a,int N)
{
    int i;
    for(i=0;i<N;++i)
    {
        a[i]=i;
    }
}

__global__ void doubleElements(int*a,int N)
{
    int indx=threadIdx.x + blockIdx.x*blockDim.x;
    int grid=gridDim.x*blockDim.x;
    for(int i=indx;i<N;i+=grid)
    {
        a[i]*=2;
    }
}

bool checkelements(int *a,int N)
{
    int i;
    for(i=0;i<N;++i)
    {
        if(a[i]!=i*2)return false;
    }
    return true;
}

int main()
{
    int N=10000;
    size_t size=N*sizeof(int);
    int *a;
    
    hipMallocManaged(&a,size);
    init(a,N);
    size_t thread_per_block=256;
    size_t number_of_block=32;
    doubleElements<<<number_of_block,thread_per_block>>>(a,N);
    hipDeviceSynchronize();
    bool b = checkelements(a,N);
    printf("%s",b?"ture":"false");
    hipFree(a);


}