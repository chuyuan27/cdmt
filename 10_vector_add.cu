
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<assert.h>

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess){
        fprintf(stderr,"CUDA Runtime Error: %s\n",hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

void initWith(float num,float *a ,int N)
{
    for(int i=0;i<N;++i)
    {
        a[i]=num;
    }
}

__global__ 
void addVectorsInto(float *result,float *a,float *b,int N)
{
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int stride = gridDim.x*blockDim.x;
    for(int i = index;i<N;i+=stride)
    {
        result[i]=a[i]+b[i];
    
    }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0;i<N;i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
    const int N=2<<20;
    size_t size= N*sizeof(float);

    float *a;
    float *b;
    float *c;
    checkCuda(hipMallocManaged(&a,size));
    checkCuda(hipMallocManaged(&b,size));
    checkCuda(hipMallocManaged(&c,size));

    initWith(3, a, N);
    initWith(4, b, N);
    initWith(0, c, N);

    size_t thread_per_block;
    thread_per_block=256;
    size_t number_of_block;
    number_of_block=(N+thread_per_block-1)/thread_per_block;

    addVectorsInto<<<number_of_block,thread_per_block>>>(c,a,b,N);
    
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());
    checkElementsAre(7,c,N);//注意顺序
    checkCuda(hipFree(a));
    checkCuda(hipFree(b));
    checkCuda(hipFree(c));


}