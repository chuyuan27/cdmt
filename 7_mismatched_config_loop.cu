
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void some_kernel(int value,int *a,int N)
{
    int i=threadIdx.x+blockIdx.x*blockDim.x;
    if(i<N)
    {
        a[i]=value;
    }
    
}

int main()
{
    int N=1000;
    int *a;
    int value=6;
    size_t size=N*sizeof(int);
    hipMallocManaged(&a,size); 
    size_t threadof_per_block=256;
    size_t block_numbers=(N+threadof_per_block-1)/threadof_per_block;

    some_kernel<<<threadof_per_block,block_numbers>>>(value,a,N);
    hipDeviceSynchronize();
    
    for(int i=0;i<N;++i)
    {
        if(a[i]!=value)
        {
            printf("false");
        }
    }
    printf("True");
    
    hipFree(a);
}