
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
    if(result != hipSuccess){
        fprintf(stderr,"Cuda Runtime Error:%s\n",hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

void matrixMulCPU(int *a,int *b,int *c,int N)
{
    int val=0;
    for(int row=0;row<N;++row)
    {
        for(int col=0;col<N;++col)
        {
            val=0;
            for(int k=0;k<N;++k)
            {
                val+=a[row*N+k]*b[col+k*N];
                c[row*N+col]=val;
            }
        }
    }
}

__global__ void matrixMulGPU(int *a,int*b,int*c,int N)
{
    int row = threadIdx.x + blockIdx.x*blockDim.x;
    int col = threadIdx.y + blockIdx.y*blockDim.y;
    if(row<N && col<N)
    {
        for(int k=0;k<N;++k)
        {
            int val = 0;
            val +=a[row*N+k]*b[col+k*N];
            c[row*N+col]=val;
        }
    }
}

void checkElements(int *a,int*b,int N)
{
    for(int row=0;row<N;++row)
    {
        for(int col=0;col<N;++col)
        {
            int val=0;
            val=a[row*N+col];
            if(b[row*N+col] != val)printf("Error:in row %d col %d",row,col);
        }
    }
}


int main()
{
    int N=2<<20;
    int *a;
    int *b;
    int *c_cpu;
    int *c_gpu;
    size_t size=N*N*sizeof(int);
    checkCuda(hipMallocManaged(&a,size));
    checkCuda(hipMallocManaged(&b,size));
    checkCuda(hipMallocManaged(&c_cpu,size));
    checkCuda(hipMallocManaged(&c_gpu,size));

    for( int row = 0; row < N; ++row )
        for( int col = 0; col < N; ++col )
        {
            a[row*N + col] = row;
            b[row*N + col] = col+2;
            c_cpu[row*N + col] = 0;
            c_gpu[row*N + col] = 0;
        }

    matrixMulCPU(a,b,c_cpu,N);
    dim3 threadperblock(16,16,1);
    dim3 numberofblocks((N+threadperblock.x-1)/threadperblock.x,(N+threadperblock.y-1)/threadperblock.y,1);

    matrixMulGPU<<<numberofblocks,threadperblock>>>(a,b,c_gpu,N);
    checkCuda(hipDeviceSynchronize());
    checkElements(c_cpu,c_gpu,N);

    checkCuda(hipFree(a));
    checkCuda(hipFree(b));
    checkCuda(hipFree(c_cpu));
    checkCuda(hipFree(c_gpu));

}